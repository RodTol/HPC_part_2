#include "headers/gpu_computation.h"

void initialise_cuda(double *A, double **dev_A, double **dev_B_col, double **dev_C,
 int *n_rows_local, int N, int n_loc, int irank) {
    int n_gpus;
    hipGetDeviceCount(&n_gpus);
    hipSetDevice(irank % n_gpus);
    // Allocate memory on the device
    hipMalloc( (void **) dev_A, n_rows_local[irank] * N * sizeof(double) );
    hipMalloc( (void **) dev_B_col, N * (n_loc + 1) * sizeof(double));
    hipMalloc( (void **) dev_C, n_rows_local[irank] * N * sizeof(double) );

    hipMemcpy(*dev_A, A, n_rows_local[irank] * N * sizeof(double), hipMemcpyHostToDevice);
}

void computation(int count, double *B_col, double *dev_A, double *dev_B_col, double *dev_C,
 int *n_row_local, int* displacement, int N, int n_loc, int irank, float *computation_Time, hipblasHandle_t handle) {
    hipMemcpy(dev_B_col, B_col, N * (n_loc + 1) * sizeof(double), hipMemcpyHostToDevice);
    
    const double alpha = 1.0, beta = 0.0;
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n_row_local[count], n_row_local[irank], N, &alpha,
     dev_B_col, n_row_local[count], dev_A, N, &beta, dev_C + displacement[count], N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    *computation_Time += time;
}