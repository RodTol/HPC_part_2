#include "headers/gpu_computation.h"

void alloc(const int* row_counts, const int n_loc, const int rank, const int N, const double *A,
                        double **d_A, double **d_B_col, double **d_C, hipblasHandle_t *handle) {

    int n_of_gpus;
    hipGetDeviceCount(&n_of_gpus);
    hipSetDevice(rank % n_of_gpus);
    hipblasCreate(handle);
    // Allocate memory on the device
    hipMalloc( (void **)d_A, row_counts[rank] * N * sizeof(double) );
    hipError_t err = hipMalloc((void **)d_B_col, N * (n_loc + 1) * sizeof(double));
    if (err != hipSuccess) printf("Error allocating memory on the device: %s\n", hipGetErrorString(err));
    hipMalloc( (void **)d_C, row_counts[rank] * N * sizeof(double) );
    // copy A to the device (C is already allocated on the gpu)
    hipError_t err1 = hipMemcpy(*d_A, A, row_counts[rank] * N * sizeof(double), hipMemcpyHostToDevice);
    if (err1 != hipSuccess) { printf("Error on copying A to d_A: %s\n", hipGetErrorString(err1)); }
}